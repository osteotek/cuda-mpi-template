#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void mult(int x, int y, int *res) {
  *res = x * y;
}

int gpu(int x, int y) {
  int *dev_res;
  int res = 0;
  hipMalloc((void**)&dev_res, sizof(int));
  mult<<<1, 1>>>(x, y, dev_res);
  hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_res);

  return res;
}
